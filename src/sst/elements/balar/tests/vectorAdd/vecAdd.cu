// Copyright 2009-2025 NTESS. Under the terms
// of Contract DE-NA0003525 with NTESS, the U.S.
// Government retains certain rights in this software.
//
// Copyright (c) 2009-2025, NTESS
// All rights reserved.
//
// Portions are copyright of other developers:
// See the file CONTRIBUTORS.TXT in the top level directory
// of the distribution for more information.
//
// This file is part of the SST software package. For license
// information, see the LICENSE file in the top level directory of the
// distribution.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(int *a, int *b, int *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 131072;

    // Host input vectors
    int *h_a;
    int *h_b;
    printf("init point h_a %p\n",h_a);
	//Host output vector
    int *h_c;

    // Device input vectors
    int *d_a;
    printf("init point d_a %p\n",d_a);
    int *d_b;
    //Device output vector
    int *d_c;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(int);

    // Allocate memory for each vector on host
    h_a = (int*)malloc(bytes);
    printf("malloc point h_a %p\n",h_a);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    printf("cuda malloc point d_a %p\n",d_a);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] =3;
        h_b[i] = 4;
    }

    // Copy host vectors to device
    printf("pre cpy point h_a %p\n",h_a);
    printf("pre cpy point d_a %p\n",d_a);
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);

    int blockSize, gridSize;

    // Number of threads in each thread block
    blockSize = 256;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);

    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );

    // Sum up vector c and print result divided by n, this should equal 1 within error
    int sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %d\n", sum/n);

    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
